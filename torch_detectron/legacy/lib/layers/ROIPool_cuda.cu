#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <ATen/ATen.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>


// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__global__ void RoIPoolFForward(const int nthreads, const T* bottom_data,
    const T spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const T* bottom_rois, T* top_data, int* argmax_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    int roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
    int roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
    int roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
    int roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    T bin_size_h = static_cast<T>(roi_height)
                       / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width)
                       / static_cast<T>(pooled_width);

    int hstart = static_cast<int>(floor(static_cast<T>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    T maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxidx = -1;
    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h * width + w;
        if (offset_bottom_data[bottom_index] > maxval) {
          maxval = offset_bottom_data[bottom_index];
          maxidx = bottom_index;
        }
      }
    }
    top_data[index] = maxval;
    argmax_data[index] = maxidx;
  }
}

template <typename T>
__global__ void RoIPoolFBackward(const int nthreads, const T* top_diff,
    const int* argmax_data, const int num_rois, const T spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    int bottom_offset = (roi_batch_ind * channels + c) * height * width;
    int top_offset    = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    T* offset_bottom_diff = bottom_diff + bottom_offset;
    const int* offset_argmax_data = argmax_data + top_offset;

    int argmax = offset_argmax_data[ph * pooled_width + pw];
    if (argmax != -1) {
      atomicAdd(
          offset_bottom_diff + argmax,
          static_cast<T>(offset_top_diff[ph * pooled_width + pw]));

    }
  }
}

std::tuple<at::Tensor, at::Tensor> ROIPool_forward_cuda(const at::Tensor& input,
                                const at::Tensor& rois,
                                const float spatial_scale,
                                const int pooled_height,
                                const int pooled_width) {
  AT_ASSERT(input.type().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERT(rois.type().is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  at::Tensor output = input.type().tensor({num_rois, channels, pooled_height, pooled_width});//.zero_();
  at::Tensor argmax = input.type().toScalarType(at::kInt).tensor({num_rois, channels, pooled_height, pooled_width}).zero_();

  auto output_size = num_rois * pooled_height * pooled_width * channels;
  hipStream_t stream = at::globalContext().getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv(output_size, 512L), 4096L));
  dim3 block(512);

  // TODO get dispatching to work
  using scalar_t = float;
  RoIPoolFForward<scalar_t><<<grid, block, 0, stream>>>(
       output_size,
       input.data<scalar_t>(),
       spatial_scale,
       channels,
       height,
       width,
       pooled_height,
       pooled_width,
       rois.data<scalar_t>(),
       output.data<scalar_t>(),
       argmax.data<int>());

  THCudaCheck(hipGetLastError());
  return std::make_tuple(output, argmax);
}

// TODO remove the dependency on input and use instead its sizes -> save memory
at::Tensor ROIPool_backward_cuda(const at::Tensor& grad,
                                 const at::Tensor& input,
                                 const at::Tensor& rois,
                                 const at::Tensor& argmax,
                                 const float spatial_scale,
                                 const int pooled_height,
                                 const int pooled_width) {
  AT_ASSERT(grad.type().is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERT(rois.type().is_cuda(), "rois must be a CUDA tensor");
  // TODO add more checks

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  at::Tensor grad_input = at::zeros_like(input);

  hipStream_t stream = at::globalContext().getCurrentCUDAStream();

  //auto input_size = input.numel();
  //dim3 grid(std::min(THCCeilDiv(input_size, 512L), 4096L));
  dim3 grid(std::min(THCCeilDiv(grad.numel(), 512L), 4096L));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() > 0) {
    // TODO get dispatching to work
    using scalar_t = float;
    RoIPoolFBackward<scalar_t><<<grid, block, 0, stream>>>(
         //input_size,
         grad.numel(),
         grad.data<scalar_t>(),
         argmax.data<int>(),
         num_rois,
         spatial_scale,
         channels,
         height,
         width,
         pooled_height,
         pooled_width,
         grad_input.data<scalar_t>(),
         rois.data<scalar_t>());
  }
  THCudaCheck(hipGetLastError());
  return grad_input;
}

